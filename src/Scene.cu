#include "hip/hip_runtime.h"
/**
 * This holds scene geometry data. ray_t casting and shading take place here.
 * @author Chris Brenton
 * @date 06/20/2011
 */

#include <iostream>
#include "Scene.h"
#include "parse/nyuparser.h"
#include "Globals.h"
#include "hit_kernel.h"
#include "structs/hitd_t.h"
#include <cutil.h>

using namespace std;

/**
 * Constructs a bounding volume heirarchy for the scene.
 */
void Scene::constructBVH()
{
}

/**
 * Reads in scene data from a file and returns a new Scene containing the newly
 * stored data.
 * @returns a pointer to the newly created Scene.
 */
Scene* Scene::read(std::fstream & input)
{
   Scene* curScene = new Scene();
   NYUParser *parser = new NYUParser;
   parser->parse(input, *curScene);
   /*
      for (int geomNdx = 0; geomNdx < (int)curScene->geometry.size(); geomNdx++)
      {
      curScene->geometry[geomNdx]->debug();
      }
    */
   delete parser;
   curScene->spheresArray = new sphere_t[curScene->spheres.size()];
   for (int i = 0; i < (int)curScene->spheres.size(); i++)
   {
      curScene->spheresArray[i] = *curScene->spheres[i];
   }
   return curScene;
}

/**
 * Checks if a ray intersects any geometry in the scene, using structs.
 * @returns true if an intersection is found.
 */
bool Scene::gpuHit(ray_t & ray, hit_t *data)
{
   /*
   // INITIALIZE closestT to MAX_DIST + 0.1
   float closestT = MAX_DIST + 0.1f;
   // INITIALIZE closestData to empty hit_t
   hit_t *closestData = new hit_t();

   // Find hit for boxes.
   // FOR each item in boxes
   for (int boxNdx = 0; boxNdx < (int)boxes.size(); boxNdx++)
   {
   float boxT = -1;
   hit_t *boxData = new hit_t();
   // IF current item is hit by ray
   if (box_hit(boxes[boxNdx], ray, &boxT, boxData) != 0)
   {
   // IF intersection is closer than closestT
   if (boxT < closestT)
   {
   // SET closestT to intersection
   closestT = boxT;
   // SET closestData to intersection data
    *closestData = *boxData;
    closestData->objIndex = boxNdx;
    }
   // ENDIF
   }
   // ENDIF
   delete boxData;
   }

   // Find hit for planes.
   // FOR each item in geometry
   for (int planeNdx = 0; planeNdx < (int)planes.size(); planeNdx++)
   {
   float planeT = -1;
   hit_t *planeData = new hit_t();
   // IF current item is hit by ray
   if (plane_hit(planes[planeNdx], ray, &planeT, planeData) != 0)
   {
   // IF intersection is closer than closestT
   if (planeT < closestT)
   {
   // SET closestT to intersection
   closestT = planeT;
   // SET closestData to intersection data
    *closestData = *planeData;
    closestData->objIndex = planeNdx;
    }
   // ENDIF
   }
   // ENDIF
   delete planeData;
   }

   // Find hit for spheres.
   // FOR each item in spheres
   for (int sphereNdx = 0; sphereNdx < (int)spheres.size(); sphereNdx++)
   {
   float sphereT = -1;
   hit_t *sphereData = new hit_t();
   // IF current item is hit by ray
   if (sphere_hit(spheres[sphereNdx], ray, &sphereT, sphereData) != 0)
   {
   // IF intersection is closer than closestT
   if (sphereT < closestT)
   {
   // SET closestT to intersection
   closestT = sphereT;
   // SET closestData to intersection data
    *closestData = *sphereData;
    closestData->objIndex = sphereNdx;
}
// ENDIF
}
// ENDIF
delete sphereData;
}

// Find hit for triangles.
// FOR each item in triangles
for (int triNdx = 0; triNdx < (int)triangles.size(); triNdx++)
{
   float triT = -1;
   hit_t *triData = new hit_t();
   // IF current item is hit by ray
   if (triangle_hit(triangles[triNdx], ray, &triT, triData) != 0)
   {
      // IF intersection is closer than closestT
      if (triT < closestT)
      {
         // SET closestT to intersection
         closestT = triT;
         // SET closestData to intersection data
         *closestData = *triData;
         closestData->objIndex = triNdx;
      }
      // ENDIF
   }
   // ENDIF
   delete triData;
}

// ENDFOR
// IF data is not null
if (data != NULL)
{
   // SET data to closestData
   *data = *closestData;
}
// ENDIF
delete closestData;
// RETURN true if closestT is less than or equal to MAX_DIST
return (closestT <= MAX_DIST);
*/
return false;
}

/**
 * Checks if a ray intersects any geometry in the scene, using Geometry.
 * @returns true if an intersection is found.
 */
bool Scene::hit(ray_t & ray, hit_t *data)
{
   // INITIALIZE closestT to MAX_DIST + 0.1
   float closestT = MAX_DIST + 0.1f;
   // INITIALIZE closestData to empty hit_t
   hit_t *closestData = new hit_t();
   // FOR each item in geometry
   for (int geomNdx = 0; geomNdx < (int)geometry.size(); geomNdx++)
   {
      float geomT = -1;
      hit_t *geomData = new hit_t();
      // IF current item is hit by ray
      if (geometry[geomNdx]->hit(ray, &geomT, geomData) != 0)
      {
         // IF intersection is closer than closestT
         if (geomT < closestT)
         {
            // SET closestT to intersection
            closestT = geomT;
            // SET closestData to intersection data
            *closestData = *geomData;
         }
         // ENDIF
      }
      // ENDIF
      delete geomData;
   }
   // ENDFOR
   // IF data is not null
   if (data != NULL)
   {
      // SET data to closestData
      *data = *closestData;
   }
   // ENDIF
   delete closestData;
   // RETURN true if closestT is less than or equal to MAX_DIST
   return (closestT <= MAX_DIST);
}

void Scene::cudaSetup(int chunkSize)
{
   cout << "Allocating device arrays...";
   // Create sphere array on device.
   spheres_size = sizeof(sphere_t) * spheres.size();
   CUDA_SAFE_CALL(hipMalloc((void**) &spheres_d, spheres_size));
   // Copy spheres to device.
   CUDA_SAFE_CALL(hipMemcpy(spheres_d, spheresArray, spheres_size,
            hipMemcpyHostToDevice));

   // Create ray array on device.
   rays_size = chunkSize * sizeof(ray_t);
   CUDA_SAFE_CALL(hipMalloc((void **) &rays_d, rays_size));

   // Create hit data array on device.
   results_size = chunkSize * sizeof(hitd_t);
   CUDA_SAFE_CALL(hipMalloc((void **) &results_d, results_size));

   // Create hit data array on host.
   results = new hitd_t[chunkSize];
   cout << "done." << endl;
}

void Scene::cudaCleanup()
{
   cout << "Cleaning up device arrays...";
   hipFree(spheres_d);
   hipFree(rays_d);
   hipFree(results_d);

   delete[] results;
   cout << "done." << endl;
}

/**
 * Casts rays into the scene and returns correctly colored pixels.
 */
Pixel* Scene::castRays(ray_t *rays, int num, int depth)
{
   Pixel *pixels = new Pixel[num];

   for (int i = 0; i < num; i++)
   {
      results[i].hit = 0;
   }
   CUDA_SAFE_CALL(hipMemcpy(results_d, results, results_size,
            hipMemcpyHostToDevice));

   // Copy rays to device.
   CUDA_SAFE_CALL(hipMemcpy(rays_d, rays, rays_size, hipMemcpyHostToDevice));
   // Calculate block size and number of blocks.
   dim3 dimGrid((int)ceil((float)num / (float)THREADS_PER_BLOCK), 1);
   dim3 dimBlock(THREADS_PER_BLOCK, 1);

   set_spheres <<< dimGrid, dimBlock >>> (spheres_d, spheres.size());
   // Test for intersection.
   cuda_hit <<< dimGrid, dimBlock >>>
      (rays_d, num, results_d);
   //cuda_hit <<< dimGrid, dimBlock >>>
      //(rays_d, num, spheres_d, spheres.size(), results_d);
   // Check for error.
   hipError_t err = hipGetLastError();
   if( hipSuccess != err)
   {
      fprintf(stderr, "Cuda error: %s: %s.\n", "kernel",
            hipGetErrorString( err) );
      exit(EXIT_FAILURE);
   }

   // Copy hit data to host.
   CUDA_SAFE_CALL(hipMemcpy(results, results_d, results_size,
            hipMemcpyDeviceToHost));

   // Color result pixels.
   for (int resultNdx = 0; resultNdx < num; resultNdx++)
   {
      hitd_t curResult = results[resultNdx];
      ray_t curRay = rays[resultNdx];
      if (curResult.hit != 0)
      {
         sphere_t hitSphere = spheresArray[curResult.objIndex];
         pixels[resultNdx] = shade(curResult, curRay);
      }
      else
      {
         pixels[resultNdx] = Pixel(0.0, 0.0, 0.0);
      }
   }

   return pixels;
}

// Calculates proper shading at the current point.
Pixel Scene::shade(hitd_t & data, ray_t & view)
{
   Pixel result(0.0, 0.0, 0.0);
   pigment_t hitP = {};
   finish_t hitF = {0};
   vec3_t hitNormal(0.0, 0.0, 0.0);
   vec3_t dataPoint = view.dir * data.t;
   dataPoint += view.point;
   box_t *b_t;
   plane_t *p_t;
   sphere_t s_t;
   triangle_t *t_t;
   switch (data.hitType) {
   case BOX_HIT:
      b_t = boxes[data.objIndex];
      hitP = b_t->p;
      hitF = b_t->f;
      //hitNormal = box_normal(b_t, data);
      break;
   case PLANE_HIT:
      p_t = planes[data.objIndex];
      hitP = p_t->p;
      hitF = p_t->f;
      //hitNormal = plane_normal(p_t);
      break;
   case SPHERE_HIT:
      s_t = spheresArray[data.objIndex];
      hitP = s_t.p;
      hitF = s_t.f;
      hitNormal = sphere_normal(s_t, dataPoint);
      break;
   case TRIANGLE_HIT:
      t_t = triangles[data.objIndex];
      hitP = t_t->p;
      hitF = t_t->f;
      //hitNormal = triangle_normal(t_t);
      break;
   default:
      cerr << "Invalid intersection type." << endl;
   }

   for (int lightNdx = 0; lightNdx < (int)lights.size(); lightNdx++)
   {
      Light *curLight = lights[lightNdx];
      // Ambient.
      result.c.r += (hitF.ambient*hitP.c.r) * curLight->r;
      result.c.g += (hitF.ambient*hitP.c.g) * curLight->g;
      result.c.b += (hitF.ambient*hitP.c.b) * curLight->b;

      // Diffuse.
      vec3_t n = hitNormal;
      n.normalize();
      vec3_t dataPoint = data.point.toHost();
      vec3_t l = curLight->location - dataPoint;
      l.normalize();
      float nDotL = n.dot(l);
      nDotL = min(nDotL, 1.0f);

      if (nDotL > 0)
      {
         result.c.r += hitF.diffuse*hitP.c.r * nDotL * curLight->r;
         result.c.g += hitF.diffuse*hitP.c.g * nDotL * curLight->g;
         result.c.b += hitF.diffuse*hitP.c.b * nDotL * curLight->b;
      }


      // Cast light feeler ray.
      /*
         ray_t feeler;
         vec3_t dataPoint = data.point.toHost();
         feeler.dir = curLight->location - dataPoint;
         feeler.dir.normalize();
         feeler.point = feeler.dir * EPSILON;
         feeler.point += dataPoint;

         hit_t tmpHit;

      // If feeler hits any object, current point is in shadow.
      bool isShadow = gpuHit(feeler, &tmpHit);

      if (!isShadow)
      {
      // Diffuse.
      vec3_t n = hitNormal;
      n.normalize();
      vec3_t l = curLight->location - dataPoint;
      l.normalize();
      float nDotL = n.dot(l);
      nDotL = min(nDotL, 1.0f);

      if (nDotL > 0)
      {
      result.c.r += hitF.diffuse*hitP.c.r * nDotL * curLight->r;
      result.c.g += hitF.diffuse*hitP.c.g * nDotL * curLight->g;
      result.c.b += hitF.diffuse*hitP.c.b * nDotL * curLight->b;
      }

      // Specular (Phong).
      vec3_t r = mReflect(l, n);
      r.normalize();
      vec3_t v = view;
      v.normalize();
      float rDotV = r.dot(v);
      rDotV = (float)pow(rDotV, 1.0f / hitF.roughness);
      rDotV = min(rDotV, 1.0f);

      if (rDotV > 0)
      {
      result.c.r += hitF.specular*hitP.c.r * rDotV * curLight->r;
      result.c.g += hitF.specular*hitP.c.g * rDotV * curLight->g;
      result.c.b += hitF.specular*hitP.c.b * rDotV * curLight->b;
      }
      }
       */
   }
   return result;
}

/*
   vec3_t Scene::reflect(vec3_t d, vec3_t n)
   {
   return n * (2 * (-d.dot(n))) + d;
   }
 */
